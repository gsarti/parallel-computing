
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N (2048*2048)
#define THREAD_PER_BLOCK 512

__global__ void reverse(int * in, int * out, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    out[index] = in[size - index - 1];
}

int main()
{
    int * in, * out;
    int * d_in, * d_out;
    int size = N * sizeof(int);
    int i;

    hipMalloc((void**)&d_in, size);
    hipMalloc((void**)&d_out, size);
    
    in = (int *)malloc(size);
    out = (int *)malloc(size);
    
    for(i = 0; i<N; ++i)
    {
	in[i] = i;	
    } 
    
    hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
    
    reverse<<< N/THREAD_PER_BLOCK, THREAD_PER_BLOCK >>>(d_in, d_out, N);
    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

    for(i=0; i<N; ++i)
    {
	if(out[i] != in[N-i-1]) {
	    printf("error\n");
	    break;
	}
    }
    if(i == N){
        printf("correct\n");
    }

    free(in); free(out);
    hipFree(d_in);
    hipFree(d_out);
    return 0;
}
